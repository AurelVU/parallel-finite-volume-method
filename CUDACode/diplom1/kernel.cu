#include "hip/hip_runtime.h"

#include <ctime>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


hipError_t solver(float** T, int n, int m);


__global__ void solverKernel(float* T, float* T_new, int n, int m, float h, float k, float dt, float dz, float Tb, float T0, float kL, float kR, float kU, float kD, float fluxR, float fluxL, float fluxU, float fluxD, float fluxLgran, float fluxRgran, float fluxUgran, float fluxDgran, float fluxC, float fluxCLgran, float fluxCRgran, float fluxCUgran, float fluxDUgran, float fluxCLUgran, float fluxCRUgran, float fluxCLDgran, float fluxDRUgran)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= 0 && i < n && j >= 0 && j < m)
    {
        double cflaxL = 0;
        double cflaxR = 0;
        double cflaxU = 0;
        double cflaxD = 0;

        if (i == 0)
            cflaxL = fluxLgran * (Tb - T[i * m + j]);
        else
            cflaxL = fluxL * (T[(i - 1) * m + j] - T[i * m + j]);
        if (i == n - 1)
            cflaxR = fluxRgran * (T0 - T[i * m + j]);
        else
            cflaxR = fluxR * (T[(i + 1) * m + j] - T[i * m + j]);

        if (j == 0)
            cflaxD = fluxDgran * (T0 - T[i * m + j]);
        else
            cflaxD = fluxD * (T[i * m + j - 1] - T[i * m + j]);
        if (j == m - 1)
            cflaxU = fluxUgran * (T0 - T[i * m + j]);
        else
            cflaxU = fluxU * (T[i * m + j + 1] - T[i * m + j]);

        T_new[i * m + j] = T[i * m + j] - dt * h * h * dz * (cflaxL + cflaxR + cflaxU + cflaxD);
    }
}

int main()
{
    int n = 210;
    int m = 210;
    float** T = new float*[n];
    for (int i = 0; i < n; i++)
    {
        T[i] = new float[m];
        for (int j = 0; j < m; j++)
            T[i][j] = 0;
    }

    clock_t start = clock();
    hipError_t cudaStatus = solver(T, n, m);
    clock_t end = clock();
    double seconds = (double)(end - start) / CLOCKS_PER_SEC;
    printf("The time: %f seconds\n", seconds);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}



hipError_t solver(float** T, int n, int m)
{
    double h = 0.1;
    double k = 50;

    double dt = 1;
    double dz = 1;
    double Tb = 240;
    double T0 = 0;

    double kL = 1;
    double kR = 0;
    double kU = 10;
    double kD = 10;

    double** T_new = new double* [n];
    for (int i = 0; i < n; i++)
        T_new[i] = new double[m];

    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            T_new[i][j] = 0;

	
    float* dev_T = 0;
    float* dev_T_new = 0;
	
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_T, n * m * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_T_new, n * m * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    dim3 gridSize = dim3((n + 31) / 32, (m + 31) / 32, 1);    //������ ������������� �����
    dim3 blockSize = dim3(32, 32, 1); //������ ������������� �����


    // Launch a kernel on the GPU with one thread for each element.
    for (int i = 0; i < 500; i++)
    {
        double fluxR = -k * h;
        double fluxL = -k * h;
        double fluxU = -k * h;
        double fluxD = -k * h;

        double fluxLgran = -kL * h * 2;
        double fluxRgran = -kR * h * 2;
        double fluxUgran = -kU * h * 2;
        double fluxDgran = -kD * h * 2;

        double fluxC = -4 * k * h;

        double fluxCLgran = fluxR + fluxLgran + fluxU + fluxD;
        double fluxCRgran = fluxRgran + fluxL + fluxU + fluxD;
        double fluxCUgran = fluxR + fluxL + fluxUgran + fluxD;
        double fluxDUgran = fluxR + fluxL + fluxU + fluxDgran;

        double fluxCLUgran = fluxR + fluxLgran + fluxUgran + fluxD;
        double fluxCRUgran = fluxRgran + fluxL + fluxUgran + fluxD;
        double fluxCLDgran = fluxR + fluxLgran + fluxU + fluxDgran;
        double fluxDRUgran = fluxRgran + fluxL + fluxU + fluxDgran;

    	
        solverKernel <<<gridSize, blockSize >>> (dev_T, dev_T_new, n, m, h, k, dt, dz, Tb, T0, kL, kR, kU, kD, fluxR, fluxL, fluxU, fluxD, fluxLgran, fluxRgran, fluxUgran, fluxDgran, fluxC, fluxCLgran, fluxCRgran, fluxCUgran, fluxDUgran, fluxCLUgran, fluxCRUgran, fluxCLDgran, fluxDRUgran);
        float* a = dev_T;
        dev_T = dev_T_new;
        dev_T_new = a;
    }
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    for (int i = 0; i < n; i++)
    {
        cudaStatus = hipMemcpy(T[i], dev_T + i * m, n * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
            printf("%.2f ", T[i][j]);
        printf("\n");
    }
    
Error:
    hipFree(dev_T);
    hipFree(dev_T_new);

    return cudaStatus;
}