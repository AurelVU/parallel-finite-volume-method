#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "CUDASolver.h"
#include "../Utils/utils.h"


__global__ void solverKernel(float* T, float* T_new, int n, int m, params param, float fluxR, float fluxL, float fluxU, float fluxD, float fluxLgran, float fluxRgran, float fluxUgran, float fluxDgran, float fluxC, float fluxCLgran, float fluxCRgran, float fluxCUgran, float fluxDUgran, float fluxCLUgran, float fluxCRUgran, float fluxCLDgran, float fluxDRUgran)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= 0 && i < n && j >= 0 && j < m)
    {
        double cflaxL = 0;
        double cflaxR = 0;
        double cflaxU = 0;
        double cflaxD = 0;

        if (i == 0)
            cflaxL = fluxLgran * (param.Tb - T[i * m + j]);
        else
            cflaxL = fluxL * (T[(i - 1) * m + j] - T[i * m + j]);
        if (i == n - 1)
            cflaxR = fluxRgran * (param.T0 - T[i * m + j]);
        else
            cflaxR = fluxR * (T[(i + 1) * m + j] - T[i * m + j]);

        if (j == 0)
            cflaxD = fluxDgran * (param.T0 - T[i * m + j]);
        else
            cflaxD = fluxD * (T[i * m + j - 1] - T[i * m + j]);
        if (j == m - 1)
            cflaxU = fluxUgran * (param.T0 - T[i * m + j]);
        else
            cflaxU = fluxU * (T[i * m + j + 1] - T[i * m + j]);

        T_new[i * m + j] = T[i * m + j] - param.dt * param.h * param.h * param.dz * (cflaxL + cflaxR + cflaxU + cflaxD);
    }
}

hipError_t internel_solver(float** T, int n, int m, int time, params param)
{
    double** T_new = new double* [n];
    for (int i = 0; i < n; i++)
        T_new[i] = new double[m];

    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            T_new[i][j] = 0;


    float* dev_T = 0;
    float* dev_T_new = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_T, n * m * sizeof(float));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_T_new, n * m * sizeof(float));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    dim3 gridSize = dim3((n + 31) / 32, (m + 31) / 32, 1);
    dim3 blockSize = dim3(32, 32, 1);

    for (int i = 0; i < time; i++)
    {
        double fluxR = -param.k * param.h;
        double fluxL = -param.k * param.h;
        double fluxU = -param.k * param.h;
        double fluxD = -param.k * param.h;

        double fluxLgran = -param.kL * param.h * 2;
        double fluxRgran = -param.kR * param.h * 2;
        double fluxUgran = -param.kU * param.h * 2;
        double fluxDgran = -param.kD * param.h * 2;

        double fluxC = -4 * param.k * param.h;

        double fluxCLgran = fluxR + fluxLgran + fluxU + fluxD;
        double fluxCRgran = fluxRgran + fluxL + fluxU + fluxD;
        double fluxCUgran = fluxR + fluxL + fluxUgran + fluxD;
        double fluxDUgran = fluxR + fluxL + fluxU + fluxDgran;

        double fluxCLUgran = fluxR + fluxLgran + fluxUgran + fluxD;
        double fluxCRUgran = fluxRgran + fluxL + fluxUgran + fluxD;
        double fluxCLDgran = fluxR + fluxLgran + fluxU + fluxDgran;
        double fluxDRUgran = fluxRgran + fluxL + fluxU + fluxDgran;


        solverKernel <<<gridSize, blockSize>>> (dev_T, dev_T_new, n, m, param, fluxR, fluxL, fluxU, fluxD, fluxLgran, fluxRgran, fluxUgran, fluxDgran, fluxC, fluxCLgran, fluxCRgran, fluxCUgran, fluxDUgran, fluxCLUgran, fluxCRUgran, fluxCLDgran, fluxDRUgran);
        float* a = dev_T;
        dev_T = dev_T_new;
        dev_T_new = a;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    for (int i = 0; i < n; i++)
    {
        cudaStatus = hipMemcpy(T[i], dev_T + i * m, n * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            goto Error;
        }
    }

Error:
    hipFree(dev_T);
    hipFree(dev_T_new);

    return cudaStatus;
}


float** cuda_solver(float** T_inp, int n, int m, int time, params param, int rank, int size, int miniN, float* left_buffer, float* right_buffer)
{
    if (rank == 0 && size == 1)
        miniN = n;
	
    float** T = T_inp;//copy2DArray<float>(T_inp, n, m);
	
    hipError_t cudaStatus = internel_solver(T, n, m, time, param);

    if (cudaStatus != hipSuccess)
        throw cudaStatus;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
        throw cudaStatus;
    return T;
}

bool isSupportedCUDA()
{
	int num_gpus = 0;
    hipError_t err = hipGetDeviceCount(&num_gpus);
    return num_gpus > 0;
}